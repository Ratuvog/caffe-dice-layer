#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include "caffe/layers/dice_layer.hpp"

namespace caffe {

template <typename Dtype>
void DiceLayer<Dtype>::Reshape(const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  vector<int> top_shape(0);
  top[0]->Reshape(top_shape);
}

template <typename Dtype>
void DiceLayer<Dtype>::Forward_cpu(const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const Dtype* label = bottom[1]->cpu_data();
  const Dtype* score = bottom[0]->cpu_data();

  const int count = bottom[1]->count();

  for(int i = 0; i < count; ++i) {
      bottom[1]->mutable_cpu_diff()[i] = score[i] >= score[i+count] ? 0 : 1;
  }
  const Dtype* prediction = bottom[1]->cpu_diff();

  label_sum = caffe_cpu_asum(count, label);

  prediction_sum = caffe_cpu_asum(count, prediction);

  Dtype *intersection = bottom[1]->mutable_cpu_diff();
  caffe_mul(count, prediction, label, intersection);

  intersection_sum = caffe_cpu_asum(count, intersection);

  top[0]->mutable_cpu_data()[0] = 2.*intersection_sum/(label_sum + prediction_sum);

}

template <typename Dtype>
__global__ void ArgMax(const int n, const Dtype* score, Dtype* predictions) {
  CUDA_KERNEL_LOOP(i, n) {
      predictions[i] = score[i] >= score[i+n] ? 0 : 1;
  }
}

template <typename Dtype>
void DiceLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  LOG(INFO) << this->type() << bottom[1]->shape_string() << " " << bottom[0]->shape_string();

  const Dtype* label = bottom[1]->gpu_data();
  const Dtype* score = bottom[0]->gpu_data();
  const int count = bottom[1]->count();

  // NOLINT_NEXT_LINE(whitespace/operators)
  ArgMax<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count,
      bottom[0]->gpu_data(),
      bottom[1]->mutable_gpu_diff()
    );
  const Dtype* prediction = bottom[1]->gpu_diff();

  caffe_gpu_asum(count, prediction, &prediction_sum);
  LOG(INFO) << this->type() << " Prediction sum: " << prediction_sum;

  caffe_gpu_asum(count, label, &label_sum);
  LOG(INFO) << this->type() << " Label sum: " << label_sum;

  Dtype *intersection = bottom[1]->mutable_gpu_diff();
  caffe_gpu_mul(count, prediction, label, intersection);

  caffe_gpu_asum(count, intersection, &intersection_sum);
  LOG(INFO) << this->type() << " Intersection sum: " << intersection_sum;

  top[0]->mutable_cpu_data()[0] = 2.*intersection_sum/(label_sum + prediction_sum);
}

template <typename Dtype>
void DiceLayer<Dtype>::Backward_cpu(const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

}


template <typename Dtype>
__global__ void DoBackward(const int labels_count, const Dtype* labels, const Dtype* predictions, Dtype* bottom_diff, const Dtype intersection, const Dtype unions) {
  CUDA_KERNEL_LOOP(j, labels_count) {
    Dtype grad = labels[j] / unions - intersection / (unions * unions);

    Dtype p0 = predictions[j];
    Dtype p1 = predictions[j + labels_count];

//    if (p0 == lablels[j])

    bottom_diff[j] = grad;
    bottom_diff[j + labels_count] = -grad;
  }
}

template <typename Dtype>
void DiceLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

  if (propagate_down[0]) {
    LOG(FATAL) << this->type() << " Layer cannot backpropagate to label inputs."                                                                                                                                                                                               ;
  }
  if (propagate_down[1]) {
    const int count = bottom[0]->count();
    // NOLINT_NEXT_LINE(whitespace/operators)
    DoBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count,
      bottom[0]->gpu_data(),
      bottom[1]->gpu_data(),
      bottom[1]->mutable_gpu_diff(),
      intersection_sum,
      label_sum + prediction_sum
    );
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(DiceLayer);
REGISTER_LAYER_CLASS(Dice);

}
